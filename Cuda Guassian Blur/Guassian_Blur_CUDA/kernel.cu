#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "SDL.h"
#include "SDL_image.h"
#undef main

#include <stdio.h>
#include <memory.h>
#include <math.h>
#include <time.h>

/// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>> Defines  <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
//Change the width of the tile here (ALSO BLOCK WIDTH)
#define TILE_WIDTH 32

/// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>  Global Variables <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
//change the gaussinan blur effect here
const int maskSize = 3; // width of the blur in pixels, must be an odd value (3, 5, 7, 9, 11, 13)
const float stdv = 20.0; // strength of the blur (1.0, 3.0, 5.0, 10.0, 20.0)?
// Change this to change the image file to be loaded. Note: needs to be a JPEG.
// file sizes are relative to their name, in order from smallest to largest the name's are
// "240p", "480p", "720p", "1080p", "1440p", "4k", "8k", "16k". 
const char* IMAGE_PATH = "4k.jpg";
// Change these to set the size of the window. The aspect ratio should match that of the image
// to be loaded otherwise it'll be distorted.
const int WINDOW_WIDTH = 1280;
const int WINDOW_HEIGHT = 720;
// Host version of the convolution mask 2D array.
float h_convMask[maskSize][maskSize];
// 2D statically allocated array in device memory containing the convolution kernel.
__constant__ float d_convMask[maskSize][maskSize];
// used in the apllying of the convolution kernel, put here in order to set it once
const int offset = (maskSize - 1) / 2; // how many x or y coordinates the convolution kernel will take you away from the central origin

/// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>> Functions <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<


/// Generate the guassian convolution kernel
// CPU runnable
// based on: https://www.codewithc.com/gaussian-filter-generation-in-c/
// Modified to be adaptable, the code is messy but the timing of CUDA vs SERIAL will not be affected by this;
// Parameters:
// Width: Dimensional width of the kernel
// Height: Dimensional height of the kernel
////
void generateGuassianKernel(int width, int height) {
	double r, s = 2.0 * stdv * stdv; 
	double sum = 0.0;   // Initialization of sun for normalization

	// Loop to generate kSize x kSize kernel
	for (int x = ((width - 1) / 2) * -1; x <= ((width - 1) / 2); x++) {
		for (int y = ((width - 1) / 2) * -1; y <= ((width - 1) / 2); y++) {
			r = sqrt(x * x + y * y);
			h_convMask[x + ((width - 1) / 2)][y + ((width - 1) / 2)] = (exp(-(r * r) / s)) / (M_PI * s); // generate using the guassian function
			sum += h_convMask[x + ((width - 1) / 2)][y + ((width - 1) / 2)];// used gor normalizing the kernel (See below...)
		}
	}
	for (int i = 0; i < maskSize; ++i) // Loop to normalize the kernel so the image doesnt get dimmer
		for (int j = 0; j < maskSize; ++j)
			h_convMask[i][j] /= sum;
}


////
// Calculate the index of an element index by x,y in a one dimensional
// flattened array (row major), where y is the major axis.
// if an index is calculated to be out of range of the array, set the index to 
// minimum or maximum depending on if the index is below or above the range.
// Parameters:
// x: x coordinate of whole pixel (not accounting for RGBA values).
// y: Y coordinate of whole pixel (not accounting for RGBA values).
// imageW, imageH: width & height of the image.
////
__host__ __device__ int get1dIndex(int width, int height, int x, int y)
{
	//check x and y boundaries
	if (x < 0) 
		x = 0;
	if (x >= width) 
		x = width - 1;
	if (y < 0) 
		y = 0;
	if (y >= height) 
		y = height - 1;

	// check index boundaries
	int i = y * width * 4 + x * 4;
	if (i < 0)
		i = 0;
	if (i > width * height * 4)
		i = ((width * height * 4) - 1);
	return i;
}

////
// CUDA kernel to convolve an image using the convolution kernel
// stored in d_convMask.
// Parameters:
// inPixels: device array holding the original image data.
// outPixels: device array where the modified image should be written.
// imageW, imageH: width & height of the image.
////
__global__ void convolveKernel(float* inPixels, float* outPixels, int imageW, int imageH) {
	int globalIdX = blockIdx.x * blockDim.x + threadIdx.x;
	int globalIdY = blockIdx.y * blockDim.y + threadIdx.y;

	if (globalIdX < imageW && globalIdY < imageH) {
		int i = globalIdX;
		int j = globalIdY;

		// Extract the red, green and blue components.
		//give the position of the r,g,b pixel in the array
		int r = get1dIndex(imageW, imageH, i, j) + 0;
		int g = get1dIndex(imageW, imageH, i, j) + 1;
		int b = get1dIndex(imageW, imageH, i, j) + 2;

		//declare the sum of each pixel colour value, the sum of the pixels around multiplied by the convolution kernels related value
		float rsum = 0.0f;
		float gsum = 0.0f;
		float bsum = 0.0f;

		//loop over guassianKernel
		// x = x of convulutionKernel (Matrix Axis)
		// y = y of convolutionKernel (Matrix Axis)
		for (int x = 0; x < maskSize; x++) {
			for (int y = 0; y < maskSize; y++) {

				//dont operate on values outside array size
				if (!(((i + (x - offset)) < 0) || ((j + (y - offset)) < 0))) { // check top and left pixel overflow
					if (!(((i + (x - offset)) > imageW) || ((j + (y - offset)) > imageH))) { // check bottom and right pixel overflow

						// Get the pixel value for the corresponding kernel value and multiply it buy the convulutionKernel value that relates to it
						rsum += (d_convMask[x][y]) * inPixels[get1dIndex(imageW, imageH, x + (i - offset), y + (j - offset)) + 0];
						gsum += (d_convMask[x][y]) * inPixels[get1dIndex(imageW, imageH, x + (i - offset), y + (j - offset)) + 1];
						bsum += (d_convMask[x][y]) * inPixels[get1dIndex(imageW, imageH, x + (i - offset), y + (j - offset)) + 2];
					}
				}
			}
		}
		//pixels that are now newly calculated now guassian smoothing has been applied
		outPixels[r] = (unsigned char)(fmaxf(0, fminf(rsum, 255.0f)));
		outPixels[g] = (unsigned char)(fmaxf(0, fminf(gsum, 255.0f)));
		outPixels[b] = (unsigned char)(fmaxf(0, fminf(bsum, 255.0f)));
	}
}

////
// GPU version of the convolution code.
// Parameters:
// inPixels: array of bytes containing the original image pixels.
// outPixels: array of bytes where the modified image should be written.
// imageW, imageH: width & height of the image.
////
void convolveImageCuda(float* inPixels, float* outPixels, int imageW, int imageH)
{
	float* d_inPixels; //device copy of inPixels.
	float* d_outPixels; //device copy of outPixels.

	//Allocate device arrays.
	hipMalloc(&d_inPixels, 4 * imageW * imageH * sizeof(float));
	hipMalloc(&d_outPixels, 4 * imageW * imageH * sizeof(float));

	//Copy input pixels to device.
	hipMemcpy(d_inPixels, inPixels, 4 * sizeof(float) * imageH * imageW, hipMemcpyHostToDevice);

	//Copy convolution mask to device.
	hipMemcpyToSymbol(HIP_SYMBOL(d_convMask), h_convMask, sizeof(float) * maskSize * maskSize);

	//Setup the size of blocks and grids kernel.
	int x = ceil((double)imageW / TILE_WIDTH); // Work out how many blocks will be needed in order to cover the whole image
	int y = ceil((double)imageH / TILE_WIDTH);// Work out how many blocks will be needed in order to cover the whole image
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1); // Create a block size based on tile size
	dim3 dimGrid(x, y, 1); // Create a grid of how many blocks are needed in order to cover the whole image

	// Run the kernel
	convolveKernel << < dimGrid, dimBlock >> > (d_inPixels, d_outPixels, imageW, imageH);


	// Copy results back to outPixels.
	hipMemcpy(outPixels, d_outPixels, 4 * sizeof(float) * imageH * imageW, hipMemcpyDeviceToHost);

	//free up memory now not needed
	hipFree(d_inPixels);
	hipFree(d_outPixels);
}

////
// CPU version of the convolution code.
// Applies the convolution kernel to each pixel (including RGBA values)
// Parameters:
// inPixels: array of bytes containing the original image pixels.
// outPixels: array of bytes where the modified image should be written.
// imageW, imageH: width & height of the image.
////
void convolveImageCPU(float* inPixels, float* outPixels, int imageW, int imageH)
{
	for (int imageX = 0; imageX < imageW; imageX++) {
		for (int imageY = 0; imageY < imageH; imageY++) {
			if (imageX < imageW && imageY < imageH) {
				int i = imageX;
				int j = imageY;

				// Extract the red, green and blue components.
				//give the position of the r,g,b pixel in the array
				int r = get1dIndex(imageW, imageH, i, j) + 0;
				int g = get1dIndex(imageW, imageH, i, j) + 1;
				int b = get1dIndex(imageW, imageH, i, j) + 2;

				//declare the sum of each pixel colour value, the sum of the pixels around multiplied by the convolution kernels related value
				float rsum = 0.0f;
				float gsum = 0.0f;
				float bsum = 0.0f;

				//loop over guassianKernel
				// x = x of convulutionKernel (Matrix Axis)
				// y = y of convolutionKernel (Matrix Axis)
				for (int x = 0; x < maskSize; x++) {
					for (int y = 0; y < maskSize; y++) {

						//dont operate on values outside array size
						if (!(((i + (x - offset)) < 0) || ((j + (y - offset)) < 0))) { // check top and left pixel overflow
							if (!(((i + (x - offset)) > imageW) || ((j + (y - offset)) > imageH))) { // check bottom and right pixel overflow

								// Get the pixel value for the corresponding kernel value and multiply it buy the convulutionKernel value that relates to it
								rsum += (h_convMask[x][y]) * inPixels[get1dIndex(imageW, imageH, x + (i - offset), y + (j - offset)) + 0];
								gsum += (h_convMask[x][y]) * inPixels[get1dIndex(imageW, imageH, x + (i - offset), y + (j - offset)) + 1];
								bsum += (h_convMask[x][y]) * inPixels[get1dIndex(imageW, imageH, x + (i - offset), y + (j - offset)) + 2];
							}
						}
					}
				}
				//pixels that are now newly calculated now guassian smoothing has been applied
				outPixels[r] = (unsigned char)(fmaxf(0, fminf(rsum, 255.0f)));
				outPixels[g] = (unsigned char)(fmaxf(0, fminf(gsum, 255.0f)));
				outPixels[b] = (unsigned char)(fmaxf(0, fminf(bsum, 255.0f)));
			}
		}
	}
}

////
// Program entry point.
////
int main(int argc, int** argv)
{
	generateGuassianKernel(maskSize, maskSize);
	// Initialize SDL and create window.
	SDL_Init(SDL_INIT_VIDEO);
	SDL_Window* window = SDL_CreateWindow(
		"Guassian Blur Applicator, CPU vs GPU",
		SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED,
		WINDOW_WIDTH, WINDOW_HEIGHT, 0);
	SDL_Renderer* renderer = SDL_CreateRenderer(
		window,
		-1,
		SDL_RENDERER_ACCELERATED | SDL_RENDERER_PRESENTVSYNC);

	// Load a photo based on image path set at the top.
	SDL_Surface* image = IMG_Load(IMAGE_PATH);
	printf("Loaded %dx%d image.\n", image->w, image->h);
	// Copy to a new surface so that we know the format (32 bit RGBA).
	SDL_Surface* surface = SDL_CreateRGBSurface(0, image->w, image->h, 32, 0x000000ff, 0x0000ff00, 0x00ff0000, 0xff000000);
	SDL_BlitSurface(image, NULL, surface, NULL);
	SDL_FreeSurface(image);
	image = NULL;
	//retreve the image size from the surface of the SDL panel
	int imageSize = surface->w * surface->h;
	
	// Allocate a pointer and space in memory for pixel data from the surface,
	// contains the RGBA values for every pixel repeeated over and over.
	// Note: stored in row major order
	float* floatPixels;
	hipHostMalloc(&floatPixels, 4 * imageSize * sizeof(float));
	float* floatPixelsOut;
	hipHostMalloc(&floatPixelsOut, 4 * imageSize * sizeof(float));
	float* floatPixelsStore;
	hipHostMalloc(&floatPixelsStore, 4 * imageSize * sizeof(float));

	// Copy surface data (image)
	unsigned char* surfacePixels = (unsigned char*)surface->pixels;
	for (int i = 0; i < imageSize; i++) {
		floatPixels[i * 4 + 0] = ((float)surfacePixels[i * 4 + 0]);
		floatPixels[i * 4 + 1] = ((float)surfacePixels[i * 4 + 1]);
		floatPixels[i * 4 + 2] = ((float)surfacePixels[i * 4 + 2]);
	}

	//CPU run and time
	clock_t CPUStart = clock();
	convolveImageCPU(floatPixels, floatPixelsOut, surface->w, surface->h);
	clock_t CPUEnd = clock();
	float CPUms = 1000.0f * (CPUEnd - CPUStart) / CLOCKS_PER_SEC;
	printf("CPU Convolution took %fms.\n\n", CPUms);

	//Store a copy of CPU out pixel ready to compare to GPU pixels
	for (int i = 0; i < imageSize; i++) {
		floatPixelsStore[i * 4 + 0] = ((float)floatPixelsOut[i * 4 + 0]);
		floatPixelsStore[i * 4 + 1] = ((float)floatPixelsOut[i * 4 + 1]);
		floatPixelsStore[i * 4 + 2] = ((float)floatPixelsOut[i * 4 + 2]);
	}

	//GPU run and time
	clock_t GPUStart = clock();
	convolveImageCuda(floatPixels, floatPixelsOut, surface->w, surface->h);
	clock_t GPUEnd = clock();
	float GPUms = 1000.0f * (GPUEnd - GPUStart) / CLOCKS_PER_SEC;
	printf("GPU Convolution took %fms.\n\n", GPUms);

	int correctPixels = 0;
	//Compare Results
	for (int i = 0; i < imageSize; i++) {
		if (floatPixelsStore[i * 4 + 0] == ((float)floatPixelsOut[i * 4 + 0])) {
			if (floatPixelsStore[i * 4 + 1] == ((float)floatPixelsOut[i * 4 + 1])) {
				if (floatPixelsStore[i * 4 + 2] == ((float)floatPixelsOut[i * 4 + 2])) {
					correctPixels++;
				}
			}
		}
	}

	float difference = ((correctPixels / imageSize) * 100);
	printf("CPU vs GPU Convolution Simularity %f Percent \n\n", difference);

	// Allocate a texture that will be the actual image drawn to the screen.
	SDL_Texture* texture = SDL_CreateTexture(
		renderer,
		SDL_PIXELFORMAT_ABGR8888,
		SDL_TEXTUREACCESS_STREAMING,
		surface->w, surface->h);

	unsigned char* pixelsTmp;
	int pitch;

	SDL_LockTexture(texture, NULL, (void**)(&pixelsTmp), &pitch);

	// put the pixels from the calulations of the convolve kernel in pixelstmp ready
	// to render the image
	for (int i = 0; i < imageSize; i++) {
		pixelsTmp[i * 4 + 0] = (unsigned char)(floatPixelsOut[i * 4]);
		pixelsTmp[i * 4 + 1] = (unsigned char)(floatPixelsOut[i * 4 + 1]);
		pixelsTmp[i * 4 + 2] = (unsigned char)(floatPixelsOut[i * 4 + 2]);
	}

	SDL_UnlockTexture(texture);

	// Draw the image.
	SDL_RenderCopy(renderer, texture, NULL, NULL);
	SDL_RenderPresent(renderer);


	// Main loop - runs continually until quit.
	bool running = true;
	while (running) {
		SDL_Event event;
		while (SDL_PollEvent(&event)) {
			if (event.type == SDL_QUIT) {
				// User pressed the "X", Alt-4F, etc...
				running = false;
			}
		}
	}

	// Main loop finished - quit.
	hipDeviceSynchronize();
	SDL_DestroyTexture(texture);
	SDL_FreeSurface(surface);
	SDL_DestroyRenderer(renderer);
	SDL_DestroyWindow(window);
	SDL_Quit();

	// Free Up Memory (D_pixels deallocated in convolveImageCuda)
	hipFree(floatPixels);
	hipFree(floatPixelsOut);
	hipFree(floatPixelsStore);
	hipFree(d_convMask);


	return 0;
}
